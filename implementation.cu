/*
============================================================================
Filename    : algorithm.c
Author      : Your name goes here
SCIPER      : Your SCIPER number
============================================================================
*/

#include <iostream>
#include <iomanip>
#include <sys/time.h>
#include <hip/hip_runtime.h>
using namespace std;

// CPU Baseline
void array_process(double *input, double *output, int length, int iterations)
{
    double *temp;

    for(int n=0; n<(int) iterations; n++)
    {
        for(int i=1; i<length-1; i++)
        {
            for(int j=1; j<length-1; j++)
            {
                output[(i)*(length)+(j)] = (input[(i-1)*(length)+(j-1)] +
                                            input[(i-1)*(length)+(j)]   +
                                            input[(i-1)*(length)+(j+1)] +
                                            input[(i)*(length)+(j-1)]   +
                                            input[(i)*(length)+(j)]     +
                                            input[(i)*(length)+(j+1)]   +
                                            input[(i+1)*(length)+(j-1)] +
                                            input[(i+1)*(length)+(j)]   +
                                            input[(i+1)*(length)+(j+1)] ) / 9;

            }
        }
        output[(length/2-1)*length+(length/2-1)] = 1000;
        output[(length/2)*length+(length/2-1)]   = 1000;
        output[(length/2-1)*length+(length/2)]   = 1000;
        output[(length/2)*length+(length/2)]     = 1000;

        temp = input;
        input = output;
        output = temp;
    }
}

__global__
void gpu_calculation(double* input, double* output, int length)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * length + x;
    if(x > 1 && x < length - 1 && y > 1 && y < length - 1) {
        output[index] = 0;/*(input[(x-1)*(length)+(y-1)] +
                        input[(x-1)*(length)+(y)]   +
                        input[(x-1)*(length)+(y+1)] +
                        input[(x)*(length)+(y-1)]   +
                        input[(x)*(length)+(y)]     +
                        input[(x)*(length)+(y+1)]   +
                        input[(x+1)*(length)+(y-1)] +
                        input[(x+1)*(length)+(y)]   +
                        input[(x+1)*(length)+(y+1)]) / 9;*/
            
    }
    
    /*if(x == length / 2 - 1 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 && y == length / 2 - 1) {
        return;
    }
    if(x == length / 2 - 1 && y == length / 2) {
        return;
    }
    if(x == length / 2 && y == length / 2) {
        return;
    }*/
    
   
}


// GPU Optimized function
void GPU_array_process(double *input, double *output, int length, int iterations)
{
    //Cuda events for calculating elapsed time
    hipEvent_t cpy_H2D_start, cpy_H2D_end, comp_start, comp_end, cpy_D2H_start, cpy_D2H_end;
    hipEventCreate(&cpy_H2D_start);
    hipEventCreate(&cpy_H2D_end);
    hipEventCreate(&cpy_D2H_start);
    hipEventCreate(&cpy_D2H_end);
    hipEventCreate(&comp_start);
    hipEventCreate(&comp_end);

    /* Preprocessing goes here */
    size_t size = length*length*sizeof(double);
    double* gpu_input;
    double* gpu_output;
    cout<<hipSuccess<<endl;
    
    hipEventRecord(cpy_H2D_start);
    /* Copying array from host to device goes here */
    cout<<hipMalloc( (void**)&gpu_input, size)<<endl;
    cout<<hipMalloc( (void**)&gpu_output, size)<<endl;
    cout<<hipMemcpy((void*)gpu_input, (void*)input, size, hipMemcpyHostToDevice)<<endl;
    cout<<hipMemcpy((void*)gpu_output, (void*)output, size, hipMemcpyHostToDevice)<<endl;
    hipDeviceSynchronize();
    
    hipEventRecord(cpy_H2D_end);
    hipEventSynchronize(cpy_H2D_end);

    //Copy array from host to device
    hipEventRecord(comp_start);
    /* GPU calculation goes here */
    int thrsPerBlock(64);
    int nBlks(length/64 + 1);

    /*for(int i = 0; i < iterations-1; i++){
        gpu_calculation <<< 2048, 2048 >>>(gpu_input, gpu_output, length);
        cudaDeviceSynchronize();

        cout<<cudaGetLastError()<<endl;
        double * temp = gpu_output;
        gpu_output = gpu_input;
        gpu_input = temp;
    }*/
    gpu_calculation <<< 2048, 2048 >>>(gpu_input, gpu_output, length);

    hipDeviceSynchronize();

    hipEventRecord(comp_end);
    hipEventSynchronize(comp_end);

    hipEventRecord(cpy_D2H_start);
    /* Copying array from device to host goes here */
    cout<<hipMemcpy((void*)output, (void*)gpu_output, size, hipMemcpyDeviceToHost)<<endl;
    hipDeviceSynchronize();

    hipEventRecord(cpy_D2H_end);
    hipEventSynchronize(cpy_D2H_end);

    /* Postprocessing goes here */
    hipFree(gpu_input);
    hipFree(gpu_output);

    float time;
    hipEventElapsedTime(&time, cpy_H2D_start, cpy_H2D_end);
    cout<<"Host to Device MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, comp_start, comp_end);
    cout<<"Computation takes "<<setprecision(4)<<time/1000<<"s"<<endl;

    hipEventElapsedTime(&time, cpy_D2H_start, cpy_D2H_end);
    cout<<"Device to Host MemCpy takes "<<setprecision(4)<<time/1000<<"s"<<endl;
}